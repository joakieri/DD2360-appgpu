#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#define ERROR_RANGE 1e-9

typedef struct {
    float3 pos;
    float3 vel;
} Particle;

__global__ void oneTimestepGPU(Particle *p) {
    const int id = blockIdx.x * blockDim.x + threadIdx.x;  
    p[id].pos.x += p[id].vel.x;
    p[id].pos.y += p[id].vel.y;
    p[id].pos.z += p[id].vel.z;
}

void oneTimestepCPU(Particle *p, int n) {
    for (int i = 0; i < n; i++) {
	    p[i].pos.x += p[i].vel.x;
	    p[i].pos.y += p[i].vel.y;
	    p[i].pos.z += p[i].vel.z;
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

int main(int argc, char *argv[]) {
    int NUM_PARTICLES = atoi(argv[1]);
    int NUM_ITERATIONS = atoi(argv[2]);
    int BLOCK_SIZE = atoi(argv[3]);
    Particle pCPU[NUM_PARTICLES];
    Particle pCPUres[NUM_PARTICLES];
    Particle pGPUres[NUM_PARTICLES];
    Particle *pGPU;
    int nBlocks;
    double iStart, iElapsCPU, iElapsGPU;
    double error;
    int nErrors;


    //printf("Particles, Iterations, Thread block size, CPU time, GPU time, Errors\n");

    nBlocks = (NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;
    srand(time(0));
    for (int i = 0; i < NUM_PARTICLES; i++) {
        pCPU[i].pos.x = rand();
        pCPU[i].pos.y = rand();
        pCPU[i].pos.z = rand();
        pCPU[i].vel.x = rand();
        pCPU[i].vel.y = rand();
        pCPU[i].vel.z = rand();
	pCPUres[i] = pCPU[i];
    }
    
    hipMalloc(&pGPU, NUM_PARTICLES * sizeof(Particle));

    // Meassure CPU performance
    iStart = cpuSecond();
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        oneTimestepCPU(pCPUres, NUM_PARTICLES);
    }
    iElapsCPU = cpuSecond() - iStart;
    
    // Meassure GPU performance
    iStart = cpuSecond();
    hipMemcpy(pGPU, pCPU, NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        oneTimestepGPU<<<nBlocks, BLOCK_SIZE>>>(pGPU);
        hipDeviceSynchronize();
    }
    hipMemcpy(pGPUres, pGPU, NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
    iElapsGPU = cpuSecond() - iStart;
    hipFree(pGPU);

    // Check the number of errors
    nErrors = 0;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        error = fabs(pCPUres[i].pos.x - pGPUres[i].pos.x);
        if (error > ERROR_RANGE) {
            nErrors++;
            continue;
        }

        error = fabs(pCPUres[i].pos.y - pGPUres[i].pos.y);
        if (error > ERROR_RANGE) {
            nErrors++;
            continue;
        }

        error = fabs(pCPUres[i].pos.z - pGPUres[i].pos.z);
        if (error > ERROR_RANGE) {
            nErrors++;
            continue;
        }
    }

    printf("%d %d %d %f %f %d\n", NUM_PARTICLES, NUM_ITERATIONS, BLOCK_SIZE, iElapsCPU, iElapsGPU, nErrors);
}
